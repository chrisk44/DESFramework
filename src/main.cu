#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <fstream>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include <cstring>

#include "des/framework.h"

/*
 * Each one of these contain a __host__ __device__ doValidate[MO][123] function
*/
#include "models/mogi/mogi.h"
#include "models/okada/okada.h"

__host__   RESULT_TYPE validate_cpuM1(DATA_TYPE* x, void* dataPtr){ return mogi::doValidateM1(x, dataPtr); }
__device__ RESULT_TYPE validate_gpuM1(DATA_TYPE* x, void* dataPtr){ return mogi::doValidateM1(x, dataPtr); }

__host__   RESULT_TYPE validate_cpuM2(DATA_TYPE* x, void* dataPtr){ return mogi::doValidateM2(x, dataPtr); }
__device__ RESULT_TYPE validate_gpuM2(DATA_TYPE* x, void* dataPtr){ return mogi::doValidateM2(x, dataPtr); }

__host__   RESULT_TYPE validate_cpuO1(DATA_TYPE* x, void* dataPtr){ return okada::doValidateO1(x, dataPtr); }
__device__ RESULT_TYPE validate_gpuO1(DATA_TYPE* x, void* dataPtr){ return okada::doValidateO1(x, dataPtr); }

__host__   RESULT_TYPE validate_cpuO2(DATA_TYPE* x, void* dataPtr){ return okada::doValidateO2(x, dataPtr); }
__device__ RESULT_TYPE validate_gpuO2(DATA_TYPE* x, void* dataPtr){ return okada::doValidateO2(x, dataPtr); }

// __host__   RESULT_TYPE validate_cpuO3(DATA_TYPE* x, void* dataPtr){ return okada::doValidateO3(x, dataPtr); }
// __device__ RESULT_TYPE validate_gpuO3(DATA_TYPE* x, void* dataPtr){ return okada::doValidateO3(x, dataPtr); }

__host__ bool toBool_cpu(RESULT_TYPE result){ return result != 0; }
__device__ bool toBool_gpu(RESULT_TYPE result){ return result != 0; }

#define RESULTS_THRESHOLD 1e-13

#define ERR_INVALID_ARG -1

std::string dataPath = "../data";
bool onlyOne = false;
int startModel = 0;
int endModel = 3;
int startGrid = 1;
int endGrid = 6;

ProcessingType processingType = PROCESSING_TYPE_GPU;
bool threadBalancing          = true;
bool slaveBalancing           = true;
bool slaveDynamicScheduling   = true;
bool cpuDynamicScheduling     = true;
bool threadBalancingAverage   = false;

unsigned long batchSize           = UINT_MAX;
float batchSizeFactor             = -1;
unsigned long slaveBatchSize      = UINT_MAX;  // 1e+07
float slaveBatchSizeFactor        = -1;
unsigned long computeBatchSize    = 20;
unsigned long cpuComputeBatchSize = 1e+04;

int blockSize  = 1024;
int gpuStreams = 8;

int slowStartLimit          = 6;
unsigned long slowStartBase = 5e+05;
int minMsForRatioAdjustment = 10;

template<typename T> T fromString(char* str);
template<> float fromString<float>(char* str){ return atof(str); }
template<> int fromString<int>(char* str){ return atoi(str); }
template<> unsigned long fromString<unsigned long>(char* str){ return atoi(str); }
template<> bool fromString<bool>(char* str){ return strcmp(str, "1")==0 || strcmp(str, "true") == 0; }
template<> std::string fromString<std::string>(char* str){ return std::string(str); }

template<typename T>
T getOrDefault(int argc, char** argv, bool* found, int* i, const char* argName, const char* argNameShort, bool hasArgument, T defaultValue){
    if(*i >= argc)
        return defaultValue;

    if(strcmp(argv[*i], argName) == 0 || strcmp(argv[*i], argNameShort) == 0){
        // If it has an argument and we have it...
        if(hasArgument && (*i + 1) < argc){
            defaultValue = fromString<T>(argv[*i+1]);
            *i += 2;
        }
        // else if it doesn't have a second argument, so just mark it as 'found'
        else if(!hasArgument){
            defaultValue = {1};
            *i += 1;
        }
        // else if it has an argument and we don't have it
        else{
            fprintf(stderr, "[E] %s requires an additional argument\n", argName);
            exit(ERR_INVALID_ARG);
        }

        *found = true;
    }

    return defaultValue;
}

void printHelp(){
    printf(
        "DES Framework Usage:\n"\
        "To run locally:    mpirun -n 2 ./parallelFramework <options>\n"
        "To run on cluster: mpirun --host localhost,localhost,remotehost1,remotehost2 ~/DESFramework/parallelFramework <options>\n\n"
        "Available options (every option takes a number as an argument. For true-false arguments use 0 or 1. -cpu, -gpu, -both don't require an argument.):\n"
        "Model/Grid selection (must be the same for every participating system):\n"
        "--data                     -d              The directory containing the data files\n"
        "--model-start              -ms             The first model to test (1-4).\n"
        "--model-end                -me             The last model to test (1-4).\n"
        "--grid-start               -gs             The first grid to test (1-6).\n"
        "--grid-end                 -ge             The last grid to test (1-6).\n"
        "--only-one                 -oo             Do only one run for each grid regardless of the time it takes.\n"
        "\n"
        "Load balancing (--thread-balancing must be the same for every system, the rest can be freely adjusted per system):\n"
        "--thread-balancing         -tb             Enables the use of HPLS in the slave level for each compute thread.\n"
        "                                           This means that for each assignment, the slave will use HPLS to calculate a ratio which will\n"
        "                                           be multiplied by the slave batch size to determine the number of elements that will be assigned to each compute thread.\n"
        "--slave-balancing          -sb             Enables the use of HPLS in the master level for each slave.\n"
        "                                           This means that for each assignment request, the master will use HPLS to calculate a ratio\n"
        "                                           which will be multiplied by the global batch size to determine the number of elements that\n"
        "                                           should be assigned to that slave.\n"
        "--slave-dynamic-balancing  -sdb            Enables dynamic scheduling in the slave level. When enabled, the slave will assign the elements dynamically to the available\n"
        "                                           resources using the slave batch size, as opposed to assigning them all at once\n"
        "--cpu-dynamic-balancing    -cdb            Enables dynamic scheduling in the compute thread level for the CPU worker thread. When enabled, the elements\n"
        "                                           that have been assigned to the CPU worker thread will be assigned dynamically to each CPU core using a CPU batch size,\n"
        "                                           as opposed to statically assigning the elements equally to the available cores.\n"
        "--thread-balancing-avg     -tba            Causes the slave-level HPLS to use the average ratio for each compute thread instead of the latest one\n. Useful when\n"
        "                                           the elements are heavily imbalanced compute-wise.\n"
        "\n"
        "Element assignment (can be defined separately for each slave, except for --batch-size which is used only by the master process):\n"
        "--batch-size               -bs             The maximum number of elements for each assignment from the master node to a slave, and the multiplier of HPLS ratios.\n"
        "--batch-size-factor        -bsf            The maximum number of elements for each assignment from the master node to a slave, and the multiplier of HPLS ratios (multiplier for total elements of grid).\n"
        "--slave-batch-size         -sbs            The maximum number of elements that a slave can assign to a compute thread at a time, and the multiplier of HPLS ratios.\n"
        "--slave-batch-size-factor  -sbsf           The maximum number of elements that a slave can assign to a compute thread at a time, and the multiplier of HPLS ratios (multiplier for total elements of grid).\n"
        "--compute-batch-size       -cbs            The number of elements that each GPU thread will compute.\n"
        "--cpu-compute-batch-size   -ccbs           The batch size for CPU dynamic scheduling.\n"
        "\n"
        "GPU parameters (can be defined separately for each slave):\n"
        "--block-size               -bls            The number of threads in each GPU block.\n"
        "--gpu-streams              -gs             The number of GPU streams to be used to dispatch work to the GPU.\n"
        "\n"
        "Slow-Start technique (used only by the master system, except for minimum time for ratio adjustment which is also used by the slaves and can be freely adjusted per system):\n"
        "--slow-start-limit         -ssl            The number of assignments that should be limited by the slow-start technique, where after each step the limit is doubled.\n"
        "--slow-start-base          -ssb            The initial number of elements for the slow-start technique which will be doubled after each step.\n"
        "--min-ms-ratio             -mmr            The minimum time in milliseconds that will be considered as valid to be used to adjust HPLS ratios.\n"
        "\n"
        "Resource selection (can be defined separately for each slave) (these don't require arguments, obviously use only one of them):\n"
        "--cpu                      -cpu            Use only the CPUs of the system\n"
        "--gpu                      -gpu            Use only the GPUs of the system\n"
        "--both                     -both           Use all CPUs and GPUs of the system\n"
    );
}

void parseArgs(int argc, char** argv){
    int i = 1;
    bool found;
    while(i < argc){
        found = false;
        dataPath   = getOrDefault(argc, argv, &found, &i,"--data",          "-d", true, dataPath);
        startModel = getOrDefault(argc, argv, &found, &i, "--model-start", "-ms", true, startModel + 1) - 1;
        endModel   = getOrDefault(argc, argv, &found, &i, "--model-end",   "-me", true, endModel + 1) - 1;
        startGrid  = getOrDefault(argc, argv, &found, &i, "--grid-start",  "-gs", true, startGrid);
        endGrid    = getOrDefault(argc, argv, &found, &i, "--grid-end",    "-ge", true, endGrid);
        onlyOne    = getOrDefault(argc, argv, &found, &i, "--only-one",    "-oo", false, onlyOne ? 1 : 0) == 1 ? true : false;

        batchSize            = getOrDefault(argc, argv, &found, &i, "--batch-size",  "-bs", true, batchSize);
        batchSizeFactor      = getOrDefault(argc, argv, &found, &i, "--batch-size-factor", "-bsf", true, batchSizeFactor);
        slaveBatchSize       = getOrDefault(argc, argv, &found, &i, "--slave-batch-size",  "-sbs", true, slaveBatchSize);
        slaveBatchSizeFactor = getOrDefault(argc, argv, &found, &i, "--slave-batch-size-factor",  "-sbsf", true, slaveBatchSizeFactor);
        computeBatchSize     = getOrDefault(argc, argv, &found, &i, "--compute-batch-size",  "-cbs", true, computeBatchSize);
        cpuComputeBatchSize  = getOrDefault(argc, argv, &found, &i, "--cpu-compute-batch-size",  "-ccbs", true, cpuComputeBatchSize);

        threadBalancing        = getOrDefault(argc, argv, &found, &i, "--thread-balancing", "-tb", true, threadBalancing ? 1 : 0) == 1 ? true : false;
        slaveBalancing         = getOrDefault(argc, argv, &found, &i, "--slave-balancing", "-sb", true, slaveBalancing ? 1 : 0) == 1 ? true : false;
        slaveDynamicScheduling = getOrDefault(argc, argv, &found, &i, "--slave-dynamic-balancing", "-sdb", true, slaveDynamicScheduling ? 1 : 0) == 1 ? true : false;
        cpuDynamicScheduling   = getOrDefault(argc, argv, &found, &i, "--cpu-dynamic-balancing", "-cdb", true, cpuDynamicScheduling ? 1 : 0) == 1 ? true : false;
        threadBalancingAverage = getOrDefault(argc, argv, &found, &i, "--thread-balancing-avg", "-tba", true, threadBalancingAverage ? 1 : 0) == 1 ? true : false;

        blockSize  = getOrDefault(argc, argv, &found, &i, "--block-size",  "-bls", true, blockSize);
        gpuStreams = getOrDefault(argc, argv, &found, &i, "--gpu-streams",  "-gs", true, gpuStreams);
        slowStartLimit = getOrDefault(argc, argv, &found, &i, "--slow-start-limit",  "-ssl", true, slowStartLimit);
        slowStartBase = getOrDefault(argc, argv, &found, &i, "--slow-start-base",  "-ssb", true, slowStartBase);
        minMsForRatioAdjustment = getOrDefault(argc, argv, &found, &i, "--min-ms-ratio",  "-mmr", true, minMsForRatioAdjustment);

        if (getOrDefault(argc, argv, &found, &i, "--cpu", "-cpu", false, false) == 1) processingType = PROCESSING_TYPE_CPU;
        if (getOrDefault(argc, argv, &found, &i, "--gpu", "-gpu", false, false) == 1) processingType = PROCESSING_TYPE_GPU;
        if (getOrDefault(argc, argv, &found, &i, "--both", "-both", false, false) == 1) processingType = PROCESSING_TYPE_BOTH;

        if (getOrDefault(argc, argv, &found, &i, "--help", "-help", false, 0) == 1){
            printHelp();
            exit(0);
        }

        if(!found && i < argc){
            printf("Unknown argument: %s\n", argv[i]);
            printHelp();
            exit(1);
            break;
        }
    }

    if(dataPath.size() == 0 || dataPath.back() != '/')
        dataPath.append("/");
}

int main(int argc, char** argv){
    const std::string modelNames[4] = {
        "mogi1",
        "mogi2",
        "okada1",
        "okada2"
    };

    // Scale factor, must be >0
    float k = 2;

    std::string displFilename;
    std::string gridFilename;
    std::string outFilename;
    bool isMaster;
    std::ifstream dispfile, gridfile;
    std::ofstream outfile;
    std::string tmp;
    int stations, dims, result, rank, commSize;
    float x, y, z, de, dn, dv, se, sn, sv;
    float low, high, step;

    Stopwatch sw;

    float finalResults[4][6];
    for(int i=0; i<4; i++)
        for(int j=0; j<6; j++)
            finalResults[i][j] = 0.0;

    parseArgs(argc, argv);

    // Initialize MPI manually
    printf("Initializing MPI\n");
    MPI_Init(nullptr, nullptr);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    isMaster = rank == 0;

    MPI_Comm_rank(MPI_COMM_WORLD, &commSize);

    // For each model...
    for(int m=startModel; m<=endModel; m++){
        if(isMaster) printf("[%d] Starting model %d/4...\n", rank, m+1);
        // Open displacements file
        displFilename = dataPath + modelNames[m] + "/displ.txt";
        dispfile.open(displFilename, std::ios::in);

        // Count stations
        stations = 0;
        while(getline(dispfile, tmp)) stations++;

        if(stations < 1){
            printf("[%d] [%s \\ N/A] Got 0 displacements. Exiting.\n", rank, modelNames[m].c_str());
            exit(2);
        }

        // Reset the file
        dispfile.close();
        dispfile.open(displFilename, std::ios::in);

        // Create the model's parameters struct (the model's input data)
        float modelDataPtr[1 + stations * (9 - (m<2 ? 0 : 1))];
        modelDataPtr[0] = (float) stations;

        // Read each station's displacement data
        float *dispPtr = &modelDataPtr[1];
        if(m < 2){
            // Mogi models have x,y,z,...
            int i = 0;
            while(dispfile >> x >> y >> z >> de >> dn >> dv >> se >> sn >> sv){
                dispPtr[0*stations + i] = x;
                dispPtr[1*stations + i] = y;
                dispPtr[2*stations + i] = z;
                dispPtr[3*stations + i] = de;
                dispPtr[4*stations + i] = dn;
                dispPtr[5*stations + i] = dv;
                dispPtr[6*stations + i] = se * k;
                dispPtr[7*stations + i] = sn * k;
                dispPtr[8*stations + i] = sv * k;

                i++;
            }
        }else{
            // Okada models have x,y,...
            int i = 0;
            while(dispfile >> x >> y >> de >> dn >> dv >> se >> sn >> sv){
                dispPtr[0*stations + i] = x;
                dispPtr[1*stations + i] = y;
                dispPtr[2*stations + i] = de;
                dispPtr[3*stations + i] = dn;
                dispPtr[4*stations + i] = dv;
                dispPtr[5*stations + i] = se * k;
                dispPtr[6*stations + i] = sn * k;
                dispPtr[7*stations + i] = sv * k;

                i++;
            }
        }

        dispfile.close();

        // For each grid...
        for(int g=startGrid; g<=endGrid; g++){
            if(isMaster) printf("[%d] Starting grid %d/6\n", rank, g);
            if(m == 3 && g > 4)
                continue;

            // Open grid file
            gridFilename = dataPath + modelNames[m] + "/grid" + std::to_string(g) + ".txt";
            gridfile.open(gridFilename, std::ios::in);

            // Count dimensions
            dims = 0;
            while(getline(gridfile, tmp)) dims++;

            // Reset the file
            gridfile.close();
            gridfile.open(gridFilename, std::ios::in);

            // Read each dimension's grid information
            std::vector<Limit> limits;
            limits.resize(dims);
            unsigned long totalElements = 1;
            {
                int i = 0;
                while(gridfile >> low >> high >> step){
                    // Create the limit (lower is inclusive, upper is exclusive)
                    high += step;
                    limits[i] = Limit{ low, high, (unsigned int) ((high-low)/step), step };
                    totalElements *= limits[i].N;
                    i++;
                }
            }

            // Close the file
            gridfile.close();

            // Create the framework's parameters struct
            ParallelFrameworkParameters parameters;
            parameters.model.D = dims;
            parameters.resultSaveType = SAVE_TYPE_LIST;
            parameters.processingType = processingType;
            parameters.output.overrideMemoryRestrictions = true;
            parameters.finalizeAfterExecution = false;
            parameters.printProgress = false;
            parameters.benchmark = false;

            parameters.model.dataPtr = (void*) modelDataPtr;
            parameters.model.dataSize = (1 + stations*(9 - (m<2 ? 0 : 1))) * sizeof(float);

            parameters.threadBalancing          = threadBalancing;
            parameters.slaveBalancing           = slaveBalancing;
            parameters.slaveDynamicScheduling   = slaveDynamicScheduling;
            parameters.cpu.dynamicScheduling = cpuDynamicScheduling;
            parameters.threadBalancingAverage   = threadBalancingAverage;

            parameters.batchSize                = batchSizeFactor > 0 ? totalElements * batchSizeFactor : batchSize;
            parameters.slaveBatchSize           = slaveBatchSizeFactor > 0 ? totalElements * slaveBatchSizeFactor : slaveBatchSize;
            parameters.gpu.computeBatchSize     = computeBatchSize;
            parameters.cpu.computeBatchSize     = cpuComputeBatchSize;

            parameters.gpu.blockSize            = blockSize;
            parameters.gpu.streams              = gpuStreams;

            parameters.slowStartLimit           = slowStartLimit;
            parameters.slowStartBase            = slowStartBase;
            parameters.minMsForRatioAdjustment  = minMsForRatioAdjustment;

            float totalTime = 0;        //msec
            int numOfRuns = 0;
            int numOfResults = -2;
            // Run at least 10 seconds, and stop after 10 runs or 2 minutes
            while(true){
                // Initialize the framework object
                ParallelFramework framework(false);
                framework.init(limits, parameters);

                // Start the computation
                sw.start();
                switch(m){
                    case 0: result = framework.run<validate_cpuM1, validate_gpuM1, toBool_cpu, toBool_gpu>(); break;
                    case 1: result = framework.run<validate_cpuM2, validate_gpuM2, toBool_cpu, toBool_gpu>(); break;
                    case 2: result = framework.run<validate_cpuO1, validate_gpuO1, toBool_cpu, toBool_gpu>(); break;
                    case 3: result = framework.run<validate_cpuO2, validate_gpuO2, toBool_cpu, toBool_gpu>(); break;
                }
                sw.stop();
                if (result != 0) {
                    printf("[%d] [%s \\ %d] Error running the computation: %d\n", rank, modelNames[m].c_str(), g, result);
                    exit(-1);
                }

                if(isMaster){
                    int size = framework.getList().size();
                    if(size != numOfResults && numOfResults != -2){
                        printf("[%s \\ %d] Number of results from run %d don't match: %d -> %d.\n",
                                        modelNames[m].c_str(), g, numOfRuns, numOfResults, size);
                    }
                    numOfResults = size;
                    // printf("[%s \\ %d] Run %d: %f ms, %d results\n", modelNames[m], g, numOfRuns, size, sw.getMsec());
                }

                totalTime += sw.getMsec();
                numOfRuns++;

                int next;
                if(isMaster){
                    if(commSize > 2 || processingType == PROCESSING_TYPE_BOTH) printf("\n");
                    if(onlyOne || (totalTime > 10 * 1000 && (numOfRuns >= 10 || totalTime >= 1 * 60 * 1000))){
                        finalResults[m][g] = totalTime/numOfRuns;
                        printf("[%s \\ %d] Time: %f ms in %d runs\n",
                                    modelNames[m].c_str(), g, totalTime/numOfRuns, numOfRuns);

                        outFilename = "results_" + modelNames[m] + "_" + std::to_string(g) + ".txt";
                        // Open file to write results
                        outfile.open(outFilename, std::ios::out | std::ios::trunc);

                        auto list = framework.getList();
                        printf("[%s \\ %d] Results: %lu\n", modelNames[m].c_str(), g, list.size());
                        for(int i=0; i<std::min((int) list.size(), 5); i++){
                            printf("[%s \\ %d] ( ", modelNames[m].c_str(), g);
                            for(auto v : list[i])
                                printf("%lf ", v);

                            printf(")\n");
                        }
                        if(list.size() > 5)
                            printf("[%s \\ %d] ...%lu more results\n", modelNames[m].c_str(), g, list.size()-5);

                        printf("\n");
                        if(g==6)
                            printf("\n");

                        for(auto point : list){
                            for(auto v : point)
                                outfile << v << " ";

                            outfile << std::endl;
                        }

                        outfile.close();
                        next = 1;
                    }else{
                        next = 0;
                    }
                }

                MPI_Bcast(&next, 1, MPI_INT, 0, MPI_COMM_WORLD);

                if(next)
                    break;
            } // end while time too short

            if(onlyOne)
                break;
        } // end for each grid

    }   // end for each model

    if(isMaster){
        printf("Final results:\n");
        for(int m=startModel; m<=endModel; m++){
            for(int g=startGrid; g<=endGrid; g++)
                printf("%f\n", finalResults[m][g]);

            printf("\n");
        }
    }

    MPI_Finalize();

    return 0;
}
