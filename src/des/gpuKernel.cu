#ifndef DES_DIRECT_COMPILATION

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "gpuKernel.h"

__device__ __constant__ char constantMemoryPtr[MAX_CONSTANT_MEMORY];

void cudaMemcpyToSymbolWrapper(const void* src, size_t count, size_t offset){
    hipMemcpyToSymbol(HIP_SYMBOL(constantMemoryPtr), src, count, offset, hipMemcpyHostToDevice);
}

#endif
