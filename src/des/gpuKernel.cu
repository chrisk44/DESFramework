#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "gpuKernel.h"
#include "utilities.h"

__constant__ char constantMemoryPtr[MAX_CONSTANT_MEMORY];

void cudaMemcpyToSymbolWrapper(const void* src, size_t count, size_t offset){
    hipMemcpyToSymbol(HIP_SYMBOL(constantMemoryPtr), src, count, offset, hipMemcpyHostToDevice);
}

