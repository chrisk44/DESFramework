#include "hip/hip_runtime.h"
#include "computeThread.h"
#include "desf.h"
#include "utilities.h"
#include "gpuKernel.h"

#include <cstring>
#include <nvml.h>
#include <stdarg.h>

ComputeThread::ComputeThread(int id, std::string name, WorkerThreadType type, DesFramework& framework, ThreadCommonData& tcd)
    : m_id(id),
      m_name(std::move(name)),
      m_type(type),
      m_framework(framework),
      m_tcd(tcd),
      m_rank(m_framework.getRank()),
      m_totalCalculatedElements(0),
      m_lastCalculatedElements(0),
      m_idleTime(0.f),
      m_activeTime(0.f)
{
    init();
    m_idleStopwatch.start();
}

ComputeThread::~ComputeThread() {
    if(m_thread.joinable())
        m_thread.join();

    finalize();
}

void ComputeThread::dispatch(size_t batchSize) {
    if(m_thread.joinable()) throw std::runtime_error("Compute thread is already running or was not joined");

    m_idleStopwatch.stop();
    m_lastIdleTime = m_idleStopwatch.getMsec();
    m_idleTime += m_lastIdleTime;
    m_thread = std::thread([this, batchSize](){
        start(batchSize);
        m_idleStopwatch.start();
    });
}

void ComputeThread::wait() {
    if(!m_thread.joinable()) throw std::runtime_error("Compute thread is not running");

    m_thread.join();
}

float ComputeThread::getUtilization() const {
    if(m_type == WorkerThreadType::GPU){
        if(m_nvml.available && m_nvml.numOfSamples > 0){
            return m_nvml.totalUtilization / m_nvml.numOfSamples;
        } else {
            return 0.f;
        }
    } else {
        return m_cpuRuntime.averageUtilization;
    }
}

void ComputeThread::log(const char *text, ...) {
    static thread_local char buf[65536];

    va_list args;
    va_start(args, text);
    vsnprintf(buf, sizeof(buf), text, args);
    va_end(args);

    printf("[%d] Compute thread %d: %s", m_rank, getId(), buf);
}

void ComputeThread::init() {
    const DesConfig& config = m_framework.getConfig();

    if (m_type == WorkerThreadType::GPU) {
        // Initialize NVML to monitor the GPU
        m_nvml.available = false;
        nvmlReturn_t result = nvmlInit();
        if (result == NVML_SUCCESS){
            m_nvml.initialized = true;

            result = nvmlDeviceGetHandleByIndex(m_id, &m_nvml.gpuHandle);
            if(result == NVML_SUCCESS){
                m_nvml.available = true;
            }else{
                log("[E] Failed to get device handle for gpu %d: %s\n", m_id, nvmlErrorString(result));
            }
        } else {
            log("[E] Failed to initialize NVML: %s\n", nvmlErrorString(result));
            m_nvml.available = false;
        }

        if(m_nvml.available){
            // Get the device's name for later
            char gpuName[NVML_DEVICE_NAME_BUFFER_SIZE];
            nvmlDeviceGetName(m_nvml.gpuHandle, gpuName, NVML_DEVICE_NAME_BUFFER_SIZE);
            m_name = gpuName;

            // Get samples to save the current timestamp
            unsigned int temp = 1;
            m_nvml.samples.resize(1);
            nvmlValueType_t sampleValType;
            // Read them one by one to avoid allocating memory for the whole buffer
            while((result = nvmlDeviceGetSamples(m_nvml.gpuHandle, NVML_GPU_UTILIZATION_SAMPLES, m_nvml.lastSeenTimeStamp, &sampleValType, &temp, m_nvml.samples.data())) == NVML_SUCCESS && temp > 0){
                m_nvml.lastSeenTimeStamp = m_nvml.samples[temp-1].timeStamp;
            }

            if (result != NVML_SUCCESS && result != NVML_ERROR_NOT_FOUND) {
                log("[E] Failed to get initial utilization samples for device: %s\n", nvmlErrorString(result));
                m_nvml.available = false;
            }
        }

        // Select gpu[id]
        hipSetDevice(m_id);

        // Calculate the max batch size for the device
        m_gpuRuntime.maxGpuBatchSize = getMaxGPUBytesForGpu(m_id);
        if(config.resultSaveType == SAVE_TYPE_ALL)
            m_gpuRuntime.maxGpuBatchSize /= sizeof(RESULT_TYPE);
        else
            m_gpuRuntime.maxGpuBatchSize /= config.model.D * sizeof(DATA_TYPE);

        // Get device's properties for shared memory
        hipGetDeviceProperties(&m_gpuRuntime.deviceProp, m_id);

        // Use constant memory for data if they fit
        m_gpuRuntime.useConstantMemoryForData = config.model.dataSize > 0 &&
            config.model.dataSize <= (MAX_CONSTANT_MEMORY - config.model.D * (sizeof(Limit) + sizeof(unsigned long long)));

        // Max use 1/4 of the available shared memory for data, the rest will be used for each thread to store their point (x) and index vector (i)
        // This seems to be worse than both global and constant memory
        m_gpuRuntime.useSharedMemoryForData = false && config.model.dataSize > 0 && !m_gpuRuntime.useConstantMemoryForData &&
                                 config.model.dataSize <= m_gpuRuntime.deviceProp.sharedMemPerBlock / 4;

        // How many bytes are left in shared memory after using it for the model's data
        m_gpuRuntime.availableSharedMemory = m_gpuRuntime.deviceProp.sharedMemPerBlock - (m_gpuRuntime.useSharedMemoryForData ? config.model.dataSize : 0);

        // How many points can fit in shared memory (for each point we need D*DATA_TYPEs (for x) and D*u_int (for indices))
        m_gpuRuntime.maxSharedPoints = m_gpuRuntime.availableSharedMemory / (config.model.D * (sizeof(DATA_TYPE) + sizeof(unsigned int)));

        #ifdef DBG_START_STOP
            if(config.printProgress){
                log("useSharedMemoryForData = %d\n", m_gpuRuntime.useSharedMemoryForData);
                log("useConstantMemoryForData = %d\n", m_gpuRuntime.useConstantMemoryForData);
                log("availableSharedMemory = %d bytes\n", m_gpuRuntime.availableSharedMemory);
                log("maxSharedPoints = %d\n", m_gpuRuntime.maxSharedPoints);
            }
        #endif

        // Create streams
        m_gpuRuntime.streams.resize(config.gpu.streams);
        for(int i=0; i<config.gpu.streams; i++){
            hipStreamCreate(&m_gpuRuntime.streams[i]);
            cce();
        }

        // Allocate memory on device
        hipMalloc(&m_gpuRuntime.deviceResults, m_gpuRuntime.allocatedElements * sizeof(RESULT_TYPE));	cce();
        hipMalloc(&m_gpuRuntime.deviceListIndexPtr, sizeof(int));							cce();
        // If we have static model data but won't use constant memory, allocate global memory for it
        if(config.model.dataSize > 0 && !m_gpuRuntime.useConstantMemoryForData){
            hipMalloc(&m_gpuRuntime.deviceDataPtr, config.model.dataSize);					cce();
        }

        #ifdef DBG_MEMORY
            log("deviceResults: %p\n", (void*) m_gpuRuntime.deviceResults);
            log("deviceListIndexPtr: %p\n", (void*) m_gpuRuntime.deviceListIndexPtr);
            log("deviceDataPtr: %p\n", (void*) m_gpuRuntime.deviceDataPtr);
        #endif

        // Copy limits, idxSteps, and constant data to device
        #ifdef DBG_MEMORY
            log("Copying limits at constant memory with offset %d\n", 0);
            log("Copying idxSteps at constant memory with offset %lu\n", config.model.D * sizeof(Limit));
        #endif
        cudaMemcpyToSymbolWrapper(
            m_framework.getLimits().data(), config.model.D * sizeof(Limit), 0);
        cce();

        cudaMemcpyToSymbolWrapper(
            m_framework.getIndexSteps().data(), config.model.D * sizeof(unsigned long long),
            config.model.D * sizeof(Limit));
        cce();

        // If we have data for the model...
        if(config.model.dataSize > 0){
            // If we can use constant memory, copy it there
            if(m_gpuRuntime.useConstantMemoryForData){
                #ifdef DBG_MEMORY
                    log("Copying data at constant memory with offset %lu\n", config.model.D * (sizeof(Limit) + sizeof(unsigned long long)));
                #endif
                cudaMemcpyToSymbolWrapper(
                    config.model.dataPtr, config.model.dataSize,
                    config.model.D * (sizeof(Limit) + sizeof(unsigned long long)));
                cce()
            }
            // else copy the data to the global memory, either to be read from there or to be copied to shared memory
            else{
                hipMemcpy(m_gpuRuntime.deviceDataPtr, config.model.dataPtr, config.model.dataSize, hipMemcpyHostToDevice);
                cce();
            }
        }
    } else {
        getCpuStats(&m_cpuRuntime.startUptime, &m_cpuRuntime.startIdleTime);
    }
}

void ComputeThread::prepareForElements(size_t numOfElements) {
    // TODO: Move this to initialization and allocate as much memory as possible
    if (m_type == WorkerThreadType::GPU && m_gpuRuntime.allocatedElements < numOfElements && m_gpuRuntime.allocatedElements < m_gpuRuntime.maxGpuBatchSize) {
        #ifdef DBG_MEMORY
            size_t prevAllocatedElements = m_gpuRuntime.allocatedElements;
        #endif

        m_gpuRuntime.allocatedElements = std::min(numOfElements, m_gpuRuntime.maxGpuBatchSize);

        #ifdef DBG_MEMORY
            log("Allocating more GPU memory (%lu -> %lu elements, %lu MB)\n", prevAllocatedElements, m_gpuRuntime.allocatedElements, (m_gpuRuntime.allocatedElements*sizeof(RESULT_TYPE)) / (1024 * 1024));
            fflush(stdout);
        #endif

        // Reallocate memory on device
        hipFree(m_gpuRuntime.deviceResults);
        cce();
        hipMalloc(&m_gpuRuntime.deviceResults, m_gpuRuntime.allocatedElements * sizeof(RESULT_TYPE));
        cce();

        #ifdef DBG_MEMORY
            log("deviceResults = %p\n", m_gpuRuntime.deviceResults);
        #endif
    }
}

AssignedWork ComputeThread::getBatch(size_t batchSize) {
    AssignedWork work;
    {
        std::lock_guard<std::mutex> lock(m_tcd.syncMutex);

        // Get the current global batch start point as our starting point
        work.startPoint = m_tcd.globalBatchStart;
        // Increment the global batch start point by our batch size
        m_tcd.globalBatchStart += batchSize;

        // Check for globalBatchStart overflow and limit it to globalLast+1 to avoid later overflows
        // If the new globalBatchStart is smaller than our local start point, the increment caused an overflow
        // If the localStart point in larger than the global last, then the elements have already been exhausted
        if(m_tcd.globalBatchStart < work.startPoint || work.startPoint > m_tcd.globalLast){
            // log("Fixing globalBatchStart from %lu to %lu\n", tcd.globalBatchStart, tcd.globalLast + 1);
            m_tcd.globalBatchStart = m_tcd.globalLast + 1;
        }
    }

    if(work.startPoint > m_tcd.globalLast){
        work.startPoint = 0;
        work.numOfElements = 0;
    } else {
        size_t last = std::min(work.startPoint + batchSize - 1 , m_tcd.globalLast);
        work.numOfElements = last - work.startPoint + 1;
    }

    return work;
}

void ComputeThread::doWorkCpu(const AssignedWork &work, RESULT_TYPE* results) {
    const auto& config= m_framework.getConfig();
    cpu_kernel(config.cpu.forwardModel, config.cpu.objective, results, m_framework.getLimits().data(), config.model.D, work.numOfElements, config.model.dataPtr, config.resultSaveType == SAVE_TYPE_ALL ? nullptr : &m_tcd.listIndex,
                    m_framework.getIndexSteps().data(), work.startPoint, config.cpu.dynamicScheduling, config.cpu.computeBatchSize);
}

void ComputeThread::doWorkGpu(const AssignedWork &work, RESULT_TYPE* results) {
    const auto& config = m_framework.getConfig();

    // Initialize the list index counter
    hipMemset(m_gpuRuntime.deviceListIndexPtr, 0, sizeof(int));

    // Divide the chunk to smaller chunks to scatter accross streams
    unsigned long elementsPerStream = work.numOfElements / config.gpu.streams;
    bool onlyOne = false;
    unsigned long skip = 0;
    if(elementsPerStream == 0){
        elementsPerStream = work.numOfElements;
        onlyOne = true;
    }

    // Queue the chunks to the streams
    for(int i=0; i<config.gpu.streams; i++){
        // Adjust elementsPerStream for last stream (= total-queued)
        if(i == config.gpu.streams - 1){
            elementsPerStream = work.numOfElements - skip;
        }else{
            elementsPerStream = std::min(elementsPerStream, work.numOfElements - skip);
        }

        // Queue the kernel in stream[i] (each GPU thread gets COMPUTE_BATCH_SIZE elements to calculate)
        int gpuThreads = (elementsPerStream + config.gpu.computeBatchSize - 1) / config.gpu.computeBatchSize;

        // Minimum of (minimum of user-defined block size and number of threads to go to this stream) and number of points that can fit in shared memory
        int blockSize = std::min(std::min(config.gpu.blockSize, gpuThreads), m_gpuRuntime.maxSharedPoints);
        int numOfBlocks = (gpuThreads + blockSize - 1) / blockSize;

        #ifdef DBG_QUEUE
            log("Queueing %lu elements in stream %d (%d gpuThreads, %d blocks, %d block size), with skip=%lu\n", elementsPerStream, i, gpuThreads, numOfBlocks, blockSize, skip);
        #endif

        // Note: Point at the start of deviceResults, because the offset (because of computeBatchSize) is calculated in the kernel
        validate_kernel<<<numOfBlocks, blockSize, m_gpuRuntime.deviceProp.sharedMemPerBlock, m_gpuRuntime.streams[i]>>>(
            config.gpu.forwardModel, config.gpu.objective,
            m_gpuRuntime.deviceResults, work.startPoint,
            config.model.D, elementsPerStream, skip, m_gpuRuntime.deviceDataPtr,
            config.model.dataSize, m_gpuRuntime.useSharedMemoryForData, m_gpuRuntime.useConstantMemoryForData,
            config.resultSaveType == SAVE_TYPE_ALL ? nullptr : m_gpuRuntime.deviceListIndexPtr,
            config.gpu.computeBatchSize
        );

        // Queue the memcpy in stream[i] only if we are saving as SAVE_TYPE_ALL (otherwise the results will be fetched at the end of the current computation)
        if(config.resultSaveType == SAVE_TYPE_ALL){
            hipMemcpyAsync(&results[skip], &m_gpuRuntime.deviceResults[skip], elementsPerStream*sizeof(RESULT_TYPE), hipMemcpyDeviceToHost, m_gpuRuntime.streams[i]);
        }

        // Increase skip
        skip += elementsPerStream;

        if(onlyOne)
            break;
    }

    // Wait for all streams to finish
    for(auto& stream : m_gpuRuntime.streams){
        hipStreamSynchronize(stream);
        cce();
    }

    // If we are saving as SAVE_TYPE_LIST, fetch the results
    if(config.resultSaveType == SAVE_TYPE_LIST){
        int gpuListIndex, globalListIndexOld;
        // Get the current list index from the GPU
        hipMemcpy(&gpuListIndex, m_gpuRuntime.deviceListIndexPtr, sizeof(int), hipMemcpyDeviceToHost);

        // Increment the global list index counter
        globalListIndexOld = __sync_fetch_and_add(&m_tcd.listIndex, gpuListIndex);

        // Get the results from the GPU
        hipMemcpy(&((DATA_TYPE*)results)[globalListIndexOld], m_gpuRuntime.deviceResults, gpuListIndex * sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    }
}

void ComputeThread::start(size_t batchSize){

    const auto& config = m_framework.getConfig();

    Stopwatch activeStopwatch;
    activeStopwatch.start();

    #ifdef DBG_TIME
        Stopwatch sw;
        float time_assign=0, time_allocation=0, time_calc=0;
        sw.start();
    #endif

    #ifdef DBG_START_STOP
        log("Woke up\n");
    #endif

    size_t numOfCalculatedElements = 0;
    while(1){
        #ifdef DBG_TIME
            sw.start();
        #endif

        Stopwatch syncStopwatch;
        syncStopwatch.start();

        AssignedWork work = getBatch(batchSize);

        syncStopwatch.stop();
        m_idleTime += syncStopwatch.getMsec();

        if(work.numOfElements == 0)
            break;

        RESULT_TYPE* localResults;
        if(config.resultSaveType == SAVE_TYPE_LIST)
            localResults = m_tcd.results;
        else
            localResults = &m_tcd.results[work.startPoint - m_tcd.globalFirst];

        #ifdef DBG_TIME
            sw.stop();
            time_assign += sw.getMsec();
        #endif

        #ifdef DBG_DATA
            log("Got %lu elements starting from %lu\n", work.numOfElements, work.startPoint);
            fflush(stdout);
        #else
            #ifdef DBG_START_STOP
                log("Running for %lu elements...\n", work.numOfElements);
                fflush(stdout);
            #endif
        #endif

        #ifdef DBG_TIME
            sw.start();
        #endif

        prepareForElements(work.numOfElements);

        #ifdef DBG_TIME
            sw.stop();
            time_allocation += sw.getMsec();
            sw.start();
        #endif

        /*****************************************************************
        ******************** Calculate the results ***********************
        ******************************************************************/
        if (m_type == WorkerThreadType::GPU) {
            doWorkGpu(work, localResults);
        } else {
            doWorkCpu(work, localResults);
        }

        numOfCalculatedElements += work.numOfElements;

        #ifdef DBG_TIME
            sw.stop();
            time_calc += sw.getMsec();
        #endif

        #ifdef DBG_RESULTS_RAW
            if(config.resultSaveType == SAVE_TYPE_ALL){
                log("Results are: ");
                for (unsigned long i = 0; i < work.numOfElements; i++) {
                    printf("%f ", ((DATA_TYPE *)localResults)[i]);
                }
                printf("\n");
            }
        #endif

        #ifdef DBG_START_STOP
            log("Finished calculation\n");
        #endif
    }

    // Stop the stopwatch
    activeStopwatch.stop();
    m_activeTime += activeStopwatch.getMsec();
    m_lastRunTime = activeStopwatch.getMsec();

    m_totalCalculatedElements += numOfCalculatedElements;
    m_lastCalculatedElements = numOfCalculatedElements;

    if(m_type == WorkerThreadType::CPU){
        float endUptime, endIdleTime;
        if(m_cpuRuntime.startUptime > 0 && m_cpuRuntime.startIdleTime > 0 && getCpuStats(&endUptime, &endIdleTime) == 0){
            m_cpuRuntime.averageUtilization = 100 - 100 * (endIdleTime - m_cpuRuntime.startIdleTime) / (endUptime - m_cpuRuntime.startUptime);

        }
    } else {
            nvmlValueType_t sampleValType;
            if(m_nvml.available){            // Get number of available samples

            unsigned int tmpSamples;
            nvmlReturn_t result = nvmlDeviceGetSamples(m_nvml.gpuHandle, NVML_GPU_UTILIZATION_SAMPLES, m_nvml.lastSeenTimeStamp, &sampleValType, &tmpSamples, NULL);
            if (result != NVML_SUCCESS && result != NVML_ERROR_NOT_FOUND) {
                log("[E1] Failed to get utilization samples for device: %s\n", nvmlErrorString(result));
            }else if(result == NVML_SUCCESS){

                // Make sure we have enough allocated memory for the new samples
                if(tmpSamples > m_nvml.samples.size()){
                    m_nvml.samples.resize(tmpSamples);
                }

                result = nvmlDeviceGetSamples(m_nvml.gpuHandle, NVML_GPU_UTILIZATION_SAMPLES, m_nvml.lastSeenTimeStamp, &sampleValType, &tmpSamples, m_nvml.samples.data());
                if (result == NVML_SUCCESS) {
                    m_nvml.numOfSamples += tmpSamples;
                    for(unsigned int i=0; i<tmpSamples; i++){
                        m_nvml.totalUtilization += m_nvml.samples[i].sampleValue.uiVal;
                    }
                }else if(result != NVML_ERROR_NOT_FOUND){
                    log("[E2] Failed to get utilization samples for device: %s\n", nvmlErrorString(result));
                }
            }
        }
        }

    #ifdef DBG_TIME
        log("Benchmark:\n");
        log("Time for assignments: %f ms\n", time_assign);
        log("Time for allocations: %f ms\n", time_allocation);
        log("Time for calcs: %f ms\n", time_calc);
    #endif

    #ifdef DBG_START_STOP
        log("Finished job\n");
    #endif
}

void ComputeThread::finalize() {
    if(m_type == WorkerThreadType::GPU) {
        // Make sure streams are finished and destroy them
        for(auto& stream : m_gpuRuntime.streams){
            hipStreamDestroy(stream);
            cce();
        }

        // Deallocate device's memory
        hipFree(m_gpuRuntime.deviceResults);			cce();
        hipFree(m_gpuRuntime.deviceListIndexPtr);		cce();
        hipFree(m_gpuRuntime.deviceDataPtr);			cce();

        if(m_nvml.initialized){
            nvmlReturn_t result = nvmlShutdown();
            if (result != NVML_SUCCESS)
                log("[E] Failed to shutdown NVML: %s\n", nvmlErrorString(result));
        }
    }
}

