#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>

#include "framework.h"

using namespace std;

ParallelFramework::ParallelFramework(Limit* limits, ParallelFrameworkParameters& parameters) {
	unsigned int i;
	valid = false;

	// Verify parameters
	if (parameters.D == 0 || parameters.D>MAX_DIMENSIONS) {
		cout << "[E] Dimension must be between 1 and " << MAX_DIMENSIONS << endl;
		return;
	}

	for (i = 0; i < parameters.D; i++) {
		if (limits[i].lowerLimit > limits[i].upperLimit) {
			cout << "[E] Limits for dimension " << i << ": Lower limit can't be higher than upper limit" << endl;
			return;
		}

		if (limits[i].N == 0) {
			cout << "[E] Limits for dimension " << i << ": N must be > 0" << endl;
			return;
		}
	}

	idxSteps = new unsigned long[parameters.D];
	idxSteps[0] = 1;
	for (i = 1; i < parameters.D; i++) {
		idxSteps[i] = idxSteps[i - 1] * limits[i-1].N;
	}

	steps = new DATA_TYPE[parameters.D];
	for (i = 0; i < parameters.D; i++) {
		steps[i] = abs(limits[i].upperLimit - limits[i].lowerLimit) / limits[i].N;
	}

	totalSent = 0;
	totalElements = (long)idxSteps[parameters.D - 1] * limits[parameters.D - 1].N;
	if(! (parameters.benchmark))
		results = new RESULT_TYPE[totalElements];		// Uninitialized
		// TODO: ^^ This really is a long story (memorywise)

	toSendVector = new unsigned long[parameters.D];
	for (i = 0; i < parameters.D; i++) {
		toSendVector[i] = 0;
	}

	this->limits = limits;
	this->parameters = &parameters;

	if (this->parameters->batchSize == 0) {
		// TODO: This really is a long story (memorywise)
		this->parameters->batchSize = totalElements;
	}

	valid = true;
}

ParallelFramework::~ParallelFramework() {
	delete [] idxSteps;
	delete [] steps;
	delete [] results;
	delete [] toSendVector;
	valid = false;
}

bool ParallelFramework::isValid() {
	return valid;
}

void ParallelFramework::masterThread(MPI_Comm& comm, int numOfProcesses) {
	int finished = 0;

	//MPI_Comm_size(comm, &numOfProcesses);

	MPI_Status status;
	int mpiSource;
	ComputeProcessStatus* processStatus = new ComputeProcessStatus[100];	// TODO: numOfProcesses might change, this should be allocated dynamically (numOfProcesses might also not be valid)
#define pstatus (processStatus[mpiSource])

	unsigned long allocatedElements = parameters->batchSize;				// Number of allocated elements for results
	RESULT_TYPE* tmpResults = new RESULT_TYPE[allocatedElements];
	unsigned long* tmpToCalculate = new unsigned long[parameters->D];
	int tmpNumOfElements;	// This needs to be int because of MPI

	#if DEBUG > 2
	printf("\nMaster: processStatus: 0x%x\n", (void*) processStatus);
	printf("Master: tmpResults: 0x%x\n", (void*) tmpResults);
	printf("Master: tmpToCalculate: 0x%x\n", (void*) tmpToCalculate);
	printf("Master: &numOfProcesses: 0x%x\n", (void*) &numOfProcesses);
	printf("Master: numOfProcesses: %d\n", numOfProcesses);
	printf("Master: &tmpNumOfElements: 0x%x\n", &tmpNumOfElements);
	printf("Master: idxSteps: 0x%x\n", (void*)idxSteps);
	printf("Master: steps: 0x%x\n", (void*) steps);
	printf("Master: results: 0x%x\n", (void*) results);
	printf("Master: toSendVector: 0x%x\n\n", (void*)toSendVector);
	#endif

	while (finished < numOfProcesses) {
		// Receive request from any worker thread
		MPI_Recv(tmpResults, allocatedElements, RESULT_MPI_TYPE, MPI_ANY_SOURCE, MPI_ANY_TAG, comm, &status);
		mpiSource = status.MPI_SOURCE;
		#if DEBUG >= 2
		cout << " Master: Received " << status.MPI_TAG << " from " << status.MPI_SOURCE << endl;
		#endif

		// Initialize process details if not initialized
		if (! (pstatus.initialized)) {
			// TODO: Add any more initializations
			pstatus.currentBatchSize = parameters->batchSize;
			pstatus.initialized = true;
		}

		if (status.MPI_TAG == TAG_READY) {
			// Receive the maximum batch size reported by the slave process
			MPI_Recv(&pstatus.maxBatchSize, 1, MPI_UNSIGNED_LONG, mpiSource, TAG_MAX_DATA_COUNT, comm, &status);

			// Get next data batch to calculate
			getDataChunk(pstatus.currentBatchSize, tmpToCalculate, &tmpNumOfElements);
			pstatus.computingIndex = getIndexFromIndices(tmpToCalculate);

			// Send data
			#if DEBUG >= 2
			cout << " Master: Sending " << tmpNumOfElements << " elements to " << mpiSource << " with index " << pstatus.computingIndex << endl;
			#endif
			#if DEBUG >= 3
			cout << " Master: Sending data to " << mpiSource << ": ";
			for (unsigned int i = 0; i < parameters->D; i++) {
				cout << tmpToCalculate[i] << " ";
			}
			cout << endl;
			#endif

			// Update details for process
			pstatus.stopwatch.start();

			MPI_Send(&tmpNumOfElements, 1, MPI_INT, mpiSource, TAG_DATA_COUNT, comm);
			MPI_Send(tmpToCalculate, parameters->D, MPI_UNSIGNED_LONG, mpiSource, TAG_DATA, comm);

			// If no more data available, source will finish
			if (tmpNumOfElements == 0) {
				#if DEBUG >= 2
				cout << " Master: Slave " << mpiSource << " finishing..." << endl;
				#endif
				finished++;
				pstatus.computingIndex = totalElements;
				pstatus.finished = true;
			}

		}else if (status.MPI_TAG == TAG_RESULTS) {
			// Save received results in this->results
			MPI_Get_count(&status, RESULT_MPI_TYPE, &tmpNumOfElements);
			#if DEBUG >= 2
			printf(" Master: Saving %ld results from slave %d to results[%ld]...\n", tmpNumOfElements, mpiSource, pstatus.computingIndex);
			#endif
			#if DEBUG >= 4
			printf(" Master: Saving tmpResults: ");
			for (int i = 0; i < tmpNumOfElements; i++) {
				//printf("%d", min(tmpResults[i], 1));
				printf("%f ", tmpResults[i]);
			}
			printf(" at %d\n", pstatus.computingIndex);
			#endif

			// Update details for process
			pstatus.jobsCompleted++;
			pstatus.elementsCalculated += tmpNumOfElements;

			pstatus.stopwatch.stop();
			float completionTime = pstatus.stopwatch.getMsec();

			if (parameters->benchmark) {
				printf("Slave %d: Benchmark: %d elements, %f ms\n", mpiSource, tmpNumOfElements, completionTime);
				fflush(stdout);
			}

			if (parameters->dynamicBatchSize) {
				// Increase batch size until we hit the max

				// Adjust pstatus.currentBatchSize: Double until SS_THRESHOLD, then increse by SS_STEP
				if (pstatus.currentBatchSize < SS_THRESHOLD) {
					pstatus.currentBatchSize = std::min((int)(2*pstatus.currentBatchSize), (int)SS_THRESHOLD);
				} else {
					pstatus.currentBatchSize += SS_STEP;
				}

				// Make sure we haven't exceded the maximum batch size set by the process
				pstatus.currentBatchSize = min(pstatus.currentBatchSize, pstatus.maxBatchSize);

				if (allocatedElements < pstatus.currentBatchSize) {
					#if DEBUG >= 2
					printf("Master: Allocating more memory (%d -> %d elements, %ld MB)\n", allocatedElements, pstatus.currentBatchSize, pstatus.currentBatchSize*sizeof(RESULT_TYPE)/(1024*1024));
					#endif

					allocatedElements = pstatus.currentBatchSize;
					tmpResults = (RESULT_TYPE*)realloc(tmpResults, allocatedElements * sizeof(RESULT_TYPE));

					#if DEBUG >= 2
					printf("Master: tmpResults: 0x%x\n", (void*)tmpResults);
					#endif
				}
			}

			if(! (parameters->benchmark))
				memcpy(&results[pstatus.computingIndex], tmpResults, tmpNumOfElements*sizeof(RESULT_TYPE));


//#if DEBUG >= 4
//			printf(" Master: results after memcpy: ");
//			for (int i = 0; i < totalElements; i++) {
//				printf("%f ", results[i]);
//			}
//			printf("\n");
//#endif
		}

		// Update numOfProcesses, in case someone else joined in (TODO: is this even possible?)
		//MPI_Comm_size(comm, &numOfProcesses);
	}

	delete[] tmpResults;
	delete[] tmpToCalculate;
	delete[] processStatus;
}

void ParallelFramework::listenerThread(MPI_Comm* parentcomm) {
	// Receive connections from other processes on the network,
	// Merge them with parentcomm
	#if DEBUG >=1
	printf("Join: joinThread started\n");
	#endif
	/*
	// TODO: Open server socket at DEFAULT_PORT

	while (true) {
		// TODO: Accept a client socket
		int clientSocket;

		MPI_Comm joinedComm;
		MPI_Comm_join(clientSocket, &joinedComm);

		MPI_Intercomm_merge(joinedComm, 0, parentcomm);
	}
	*/
	#if DEBUG >=1
	printf("Join: joinThread stopped\n");
	#endif
}

void ParallelFramework::getDataChunk(unsigned long batchSize, unsigned long* toCalculate, int* numOfElements) {
	if (totalSent >= totalElements) {
		*numOfElements = 0;
		return;
	}

	if (totalElements - totalSent < batchSize)
		batchSize = totalElements - totalSent;

	// Copy toSendVector to the output
	memcpy(toCalculate, toSendVector, parameters->D * sizeof(long));
	*numOfElements = batchSize;

	unsigned int i;
	unsigned int newIndex;
	unsigned int carry = batchSize;

	for (i = 0; i < parameters->D; i++) {
		newIndex = (toSendVector[i] + carry) % limits[i].N;
		carry = (toSendVector[i] + carry) / limits[i].N;

		toSendVector[i] = newIndex;
	}

	totalSent += batchSize;
}

RESULT_TYPE* ParallelFramework::getResults() {
	return results;
}
void ParallelFramework::getIndicesFromPoint(DATA_TYPE* point, unsigned long* dst) {
	unsigned int i;

	for (i = 0; i < parameters->D; i++) {
		if (point[i] < limits[i].lowerLimit || point[i] >= limits[i].upperLimit) {
			cout << "Result query for out-of-bounds point" << endl;
			return;
		}

		// Calculate the steps for dimension i
		dst[i] = (int) round(abs(limits[i].lowerLimit - point[i]) / steps[i]);		// TODO: 1.9999997 will round to 2, verify correctness
	}

//#if DEBUG >= 4
//	cout << "Index for point ( ";
//	for (i = 0; i < parameters->D; i++)
//		cout << point[i] << " ";
//	cout << "): ";
//
//	for (i = 0; i < parameters->D; i++) {
//		cout << dst[i] << " ";
//	}
//	cout << endl;
//#endif
}
long ParallelFramework::getIndexFromIndices(unsigned long* pointIdx) {
	unsigned int i;
	long index = 0;

	for (i = 0; i < parameters->D; i++) {
		// Increase index by i*(index-steps for this dimension)
		index += pointIdx[i] * idxSteps[i];
	}

//#if DEBUG >= 4
//	cout << "Index for point ( ";
//	for (i = 0; i < parameters->D; i++)
//		cout << pointIdx[i] << " ";
//	cout << "): " << index << endl;
//#endif

	return index;
}
