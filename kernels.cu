#include "hip/hip_runtime.h"
#ifndef KERNELS_CU
#define KERNELS_CU

#include <hip/hip_runtime.h>

#include "utilities.h"

#include <iostream>
using namespace std;

// CUDA kernel to create the 'Model' object on device
template<class ImplementedModel>
__global__ void create_model_kernel(ImplementedModel** deviceModelAddress) {
	(*deviceModelAddress) = new ImplementedModel();
}

// CUDA kernel to delete the 'Model' object on device
template<class ImplementedModel>
__global__ void delete_model_kernel(ImplementedModel** deviceModelAddress) {
	delete (*deviceModelAddress);
}

// CUDA kernel to run the computation
template<class ImplementedModel>
__global__ void validate_kernel(ImplementedModel** model, unsigned long* startingPointIdx, RESULT_TYPE* results, Limit* limits, unsigned int D, unsigned int numOfElements) {
	unsigned int threadX = (blockIdx.x * BLOCK_SIZE) + threadIdx.x;
	if (threadX < numOfElements) {
		DATA_TYPE point[MAX_DIMENSIONS];
		unsigned long tmpIndex, carry;
		unsigned int i;
		/*DATA_TYPE step[MAX_DIMENSINS];
		for (i = 0; i < D; i++) {
			step[i] = abs(limits[i].lowerLimit - limits[i].upperLimit) / limits[i].N;
		}*/

		// Calculate 'myIndex = startingPointIdx + threadIdx.x' and then the exact point
		carry = threadX;
		for (i = 0; i < D; i++) {
			tmpIndex = (startingPointIdx[i] + carry) % limits[i].N;
			carry = (startingPointIdx[i] + carry) / limits[i].N;

			// Calculate the exact coordinate i
			point[i] = limits[i].lowerLimit + tmpIndex * (abs(limits[i].lowerLimit - limits[i].upperLimit) / limits[i].N);
		}

		// Run the validation function and save the result to the global memory
		results[threadX] = (*model)->validate_gpu(point);
	}
}

// CPU kernel to run the computation
template<class ImplementedModel>
void cpu_kernel(unsigned long* startingPointIdx, RESULT_TYPE* results, Limit* limits, unsigned int D, int numOfElements) {
	DATA_TYPE* point = new DATA_TYPE[D];
	DATA_TYPE* step = new DATA_TYPE[D];
	unsigned long tmpIndex;

	ImplementedModel model = ImplementedModel();

	for (unsigned int i = 0; i < D; i++) {
		step[i] = abs(limits[i].lowerLimit - limits[i].upperLimit) / limits[i].N;
	}

	for (long j = 0; j < numOfElements; j++) {
		// Calculate 'myIndex = startingPointIdx + j' and then the exact point
		unsigned int i;
		unsigned long carry = j;
		for (i = 0; i < D; i++) {
			tmpIndex = (startingPointIdx[i] + carry) % limits[i].N;
			carry = (startingPointIdx[i] + carry) / limits[i].N;

			// Calculate the exact coordinate i
			point[i] = limits[i].lowerLimit + tmpIndex * step[i];
		}

		// Run the validation function
		results[j] = model.validate_cpu(point);
#if DEBUG >=4
		//cout << "Point (" << point[0] << "," << point[1] << ") returned " << results[j] << endl;
#endif
	}

	delete[] point;
}

#endif
