#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdlib>
#include "model.cu"
#include "framework.h"

using namespace std;

class MyModel : public Model{
public:
    __host__ bool validate_cpu(float *point){
        return point[0] >= 0 && point[1] >= 0;
    }

    __device__ bool validate_gpu(float *point){
        return point[0] >= 0 && point[1] >= 0;
    }

    bool toBool(){ return true; }
};

int main(){
    int result;
    MyModel model;
    ParallelFrameworkParameters parameters;
    Limit limits[2];

    // Create a model object
    model = MyModel();

    // Create the parameters struct
    parameters.D = 2;
    parameters.batchSize = 1;
    parameters.computeBatchSize = 1;

    // Create the limits for each dimension (lower is inclusive, upper may be exclusive, depending on the step
    // TODO: Change this to provide N instead of steps
    limits[0] = Limit { -10, 10, 1 };
    limits[1] = Limit { -10, 10, 2 };

    // Declare the framework object
    ParallelFramework framework;
     
    // Initialize the framework object
    result = framework.init(limits, parameters, model);
    if (result != 0) {
        cout << "Error initializing framework: " << result << endl;
    }

    // Start the computation
    result = framework.run();
    if (result != 0) {
        cout << "Error running the computation: " << result << endl;
    }

    float point[2] = { -10, -10 };
    bool r1 = framework.getResultAt(point);
    point[0] = 9.8;
    point[1] = 9.8;
    bool r2 = framework.getResultAt(point);

    return 0;
}
